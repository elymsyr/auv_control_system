#include "hip/hip_runtime.h"
#include "EnvironmentMap.h"
#include <hip/hip_runtime.h>
#include <cstdint>

__host__ void EnvironmentMap::initialize(int w, int h) {
    width = w;
    height = h;
    x_ = y_ = yaw_ = 0.0f;
    size_t size = w * h * sizeof(uint8_t);
    hipMallocManaged(&grid, size);  // Unified Memory
    hipMallocManaged(&tempGrid, size);
    hipMemset(grid, 0, size);
}

__host__ void EnvironmentMap::cleanup() {
    hipFree(grid);
    hipFree(tempGrid);
}

__host__ EnvironmentMap::EnvironmentMap(int w, int h) : width(w), height(h) {
    size_t size = w * h * sizeof(uint8_t);  // Size for uint8_t
    hipMalloc(&grid, size);
    hipMalloc(&tempGrid, size);
    hipMemset(grid, 0, size);  // Initialize to 0
}

__host__ EnvironmentMap::~EnvironmentMap() {
    hipFree(grid);
    hipFree(tempGrid);
}

__host__ void EnvironmentMap::applyBatchUpdate(const PointBatch& batch) {
    const int blockSize = 256;
    const int gridSize = (batch.count + blockSize - 1) / blockSize;
    
    ultraFastUpdateKernel<<<gridSize, blockSize>>>(this, batch);
    hipDeviceSynchronize();
}

__device__ void EnvironmentMap::iterate(float dx, float dy) {
    x_ += dx;
    y_ += dy;
    sx_ = static_cast<int>(x_ / 25.0f);
    sy_ = static_cast<int>(y_ / 25.0f);
    x_ -= sx_ * 25.0f;
    y_ -= sy_ * 25.0f;
}

__device__ void EnvironmentMap::slideGrid() {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= width || ty >= height) return;

    int srcX = tx - sx_;
    int srcY = ty - sy_;
    int dstIdx = ty * width + tx;

    if (srcX >= 0 && srcX < width && srcY >= 0 && srcY < height) {
        int srcIdx = srcY * width + srcX;
        tempGrid[dstIdx] = grid[srcIdx];
    } else {
        tempGrid[dstIdx] = 0.0f;
    }

    __syncthreads();
    grid[dstIdx] = tempGrid[dstIdx];
}

__device__ void EnvironmentMap::setPoint(int x, int y, uint8_t value) {
    if (x >= 0 && x < width && y >= 0 && y < height) {
        grid[y * width + x] = value;
    }
}

__global__ void setPointKernel(EnvironmentMap* map, int x, int y, uint8_t value) {
    map->setPoint(x, y, value);
}

__device__ void EnvironmentMap::iterate(float a, float b, float c, float d) {}

// Kernel wrapper
__global__ void iterateMovementKernel(EnvironmentMap* map, float dx, float dy) {
    map->iterate(dx, dy);
}

__global__ void slideGridKernel(EnvironmentMap* map, int sx_, int sy_) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= map->width || ty >= map->height) return;

    // Phase 1: Copy grid -> tempGrid with shift
    int srcX = tx - sx_;
    int srcY = ty - sy_;
    int dstIdx = ty * map->width + tx;

    if (srcX >= 0 && srcX < map->width && srcY >= 0 && srcY < map->height) {
        map->tempGrid[dstIdx] = map->grid[srcY * map->width + srcX];
    } else {
        map->tempGrid[dstIdx] = 0;  // Use 0 for uint8_t
    }

    // Phase 2: Copy tempGrid -> grid (after all threads complete phase 1)
    __syncthreads();  // Now safe within block
    map->grid[dstIdx] = map->tempGrid[dstIdx];
}

__global__ void ultraFastUpdateKernel(EnvironmentMap* map, PointBatch batch) {
    const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= batch.count) return;
    
    const int2 coord = batch.coords_dev[tid];
    const uint8_t val = batch.values_dev[tid];
    
    if(coord.x >=0 && coord.x < map->width && coord.y >=0 && coord.y < map->height) {
        map->grid[coord.y * map->width + coord.x] = val;
    }
}
