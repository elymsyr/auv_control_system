#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include "environment.h"
#include <fstream>

void EnvironmentMap::initializeGrid() {
    dim3 block(16, 16);
    dim3 grid((width_ + block.x - 1) / block.x, 
              (height_ + block.y - 1) / block.y);

    initKernel<<<grid, block>>>(node_grid_, width_, height_);
    CHECK_CUDA(hipDeviceSynchronize());
}

Path EnvironmentMap::findPath(int goal_x, int goal_y) {
    int start_x = width_ / 2;
    int start_y = height_ / 2;

    dim3 block(16, 16);
    dim3 grid((width_ + 15) / 16, (height_ + 15) / 16);

    // Step 1: Reset grid
    resetGridKernel<<<grid, block>>>(node_grid_, grid_, width_, height_, goal_x, goal_y);
    CUDA_CALL(hipDeviceSynchronize());

    // Step 2: Wavefront propagation
    int* d_updated;
    int h_updated = 1;
    CUDA_CALL(hipMalloc(&d_updated, sizeof(int)));
    
    for (int iter = 0; iter < max_iter_ && h_updated; iter++) {
        h_updated = 0;
        CUDA_CALL(hipMemcpy(d_updated, &h_updated, sizeof(int), hipMemcpyHostToDevice));
        
        wavefrontKernel<<<grid, block>>>(node_grid_, width_, height_, d_updated);
        CUDA_CALL(hipMemcpy(&h_updated, d_updated, sizeof(int), hipMemcpyDeviceToHost));
    }

    // Step 3: Reconstruct path
    int2* d_path;
    int* d_path_length;
    CUDA_CALL(hipMalloc(&d_path, width_ * height_ * sizeof(int2)));
    CUDA_CALL(hipMalloc(&d_path_length, sizeof(int)));
    CUDA_CALL(hipMemset(d_path_length, 0, sizeof(int)));
    
    reconstructPathKernel<<<1, 1>>>(node_grid_, d_path, d_path_length, 
                                   start_x, start_y, goal_x, goal_y, width_);
    
    int path_length;
    CUDA_CALL(hipMemcpy(&path_length, d_path_length, sizeof(int), hipMemcpyDeviceToHost));
    
    int2* h_path = new int2[path_length];
    CUDA_CALL(hipMemcpy(h_path, d_path, path_length * sizeof(int2), hipMemcpyDeviceToHost));

    // Reverse path (start->goal)
    // std::reverse(h_path, h_path + path_length);

    // Cleanup
    CUDA_CALL(hipFree(d_updated));
    CUDA_CALL(hipFree(d_path));
    CUDA_CALL(hipFree(d_path_length));
    
    return {h_path, path_length};
}